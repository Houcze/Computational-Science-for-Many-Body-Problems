#include "hip/hip_runtime.h"
/*
 coding: utf-8

  Exercise code for Monte Calro simulation of the 2d Ising model
 This module is for simulation of 2d Ising model on the square lattice, whose Hamiltonian is given by
 $$ \mathcal{H} = -J \sum_{\langle i,j\rangle} S_i S_j - h \sum_i S_i ,$$
 where $S_i = \pm 1$.

 You can select three simulation algorithms:
 * metropolis
 * heatbath
 * cluster (Swendsen-Wang)

 The outputs are:
 * Energy: $\langle E\rangle = \langle \mathcal{H}\rangle/N$.
 * Squared magnetization: $\langle M^2\rangle = \langle (\sum_i S_i)^2\rangle/N^2$.
 * Specific heat: $N(\langle E^2\rangle - \langle E\rangle^2)/T$
 * Magnetic susceptibility: $N(\langle M^2\rangle\rangle)/T$
 * Anothor Magnetic susceptibility: $N(\langle M^2\rangle - \langle |M|^2\rangle)/T$
 * Binder ratio: $(\langle M^4\rangle/\langle M^2\rangle)/T$
*/
#include <cmath>
#include <random>
#include <hiprand.h>
#include <iostream>
#include <algorithm>
#include <fstream>

static const int threadsperblock = 256;

__global__ void metropolis_k(int *S, double *exps, int1 L, double *ran)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < L.x && iy < L.x)
    {
        int N = L.x * L.x;
        int hm = (S[ix * L.x + iy] * (S[((ix + 1) % L.x) * L.x + iy] + S[ix * L.x + (iy + 1) % L.x] + S[((ix - 1 + L.x) % L.x) * L.x + iy] + S[ix * L.x + ((iy - 1 + L.x) % L.x)]) + 4) / 2;
        if (ran[ix * L.x + iy] < exps[hm * 2 + (S[ix * L.x + iy] + 1) / 2])
        {
            S[ix * L.x + iy] *= -1;
        }
    }
}

int metropolis(int *S, double *exps, int L, int seed)
{
    double *ran;
    hipMalloc(&ran, L * L * sizeof(double));
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniformDouble(gen, ran, L * L);
    hiprandDestroyGenerator(gen);
    metropolis_k<<<ceil(L * L / threadsperblock), threadsperblock>>>(S, exps, make_int1(L), ran);
    hipFree(ran);
    return EXIT_SUCCESS;
}

void Initialize(int seed, int L, int *S)
{
    std::mt19937 generator(seed);
    std::uniform_int_distribution<int> distribution(0, 1);
    std::vector<int> S_(L * L);
    std::generate(S_.begin(), S_.end(), [&](){ return -2 * distribution(generator) + 1; });
    std::copy(S_.begin(), S_.end(), S);
}

double Calc_local_energy(int *S, int L)
{
    double local_ene = 0.0;
    for (int ix = 0; ix < L; ix++)
    {
        for (int iy = 0; iy < L; iy++)
        {
            local_ene += S[ix * L + iy] * (S[(ix + 1) % L * L + iy] + S[ix * L + (iy + 1) % L]);
        }
    }
    return local_ene;
}

template <class T>
T sum(T *arr, int N)
{
    T result{0};
    for (int i = 0; i < N; i++)
    {
        result += arr[i];
    }
    return result;
}

void MC(int L, double T, double h, int thermalization, double *mag, double *mag2, double *mag4, double *mag_abs, double *ene, double *ene2, int observation, int seed = 11)
{
    int N = L * L;
    int *S;
    S = (int *)malloc(N * sizeof(int));
    Initialize(seed, L, S);

    double *exps;
    exps = (double *)malloc(5 * 2 * sizeof(double));

    double hm;
    double sh;
    for (int i = 0; i < 5; i++)
    {
        hm = -4.0 + 2.0 * i;
        for (int j = 0; j < 2; j++)
        {
            sh = h * (2 * j - 1);
            exps[i * 2 + j] = exp(-2.0 * (hm + sh) / T);
        }
    }

    double *exps_dev;
    int *S_dev;

    hipMalloc(&exps_dev, 5 * 2 * sizeof(double));
    hipMalloc(&S_dev, L * L * sizeof(int));

    hipMemcpy(exps_dev, exps, 5 * 2 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(S_dev, S, L * L * sizeof(int), hipMemcpyHostToDevice);
    int seed_;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dis;
    for (int i = 0; i < thermalization; i++)
    {
        seed_ = dis(gen);
        metropolis(S_dev, exps_dev, L, seed_);
    }

    double local_mag;
    double local_ene;

    for (int i = 0; i < observation; i++)
    {
        seed_ = dis(gen);
        metropolis(S_dev, exps_dev, L, seed_);
        hipMemcpy(S, S_dev, L * L * sizeof(int), hipMemcpyDeviceToHost);
        local_mag = double(sum<int>(S, N)) / N;
        // std::cout << "local_mag is " << local_mag << std::endl;
        mag[i] = local_mag;
        mag2[i] = local_mag * local_mag;
        mag4[i] = local_mag * local_mag * local_mag * local_mag;
        mag_abs[i] = abs(local_mag);
        local_ene = -Calc_local_energy(S, L) / N - h * local_mag;
        // std::cout << "local_ene is " << local_ene << std::endl;
        ene[i] = local_ene;
        ene2[i] = local_ene * local_ene;
    }
    hipFree(exps_dev);
    hipFree(S_dev);
    free(exps);
}

void variance(double *e, double *e2, double *result, int N)
{
    for (int i = 0; i < N; i++)
    {
        result[i] = e2[i] - e[i] * e[i];
    }
}

void variance(int *e, int *e2, int *result, int N)
{
    for (int i = 0; i < N; i++)
    {
        result[i] = e2[i] - e[i] * e[i];
    }
}

void binder(double *m2, double *m4, double *result, int N)
{
    for (int i = 0; i < N; i++)
    {
        result[i] = m4[i] / (m2[i] * m2[i]);
    }
}

void make_bin(double *data, int bin_size_in, int data_size, double *bin_data, int *bin_size_out)
{
    *bin_size_out = bin_size_in;
    int bin_num = data_size / (*bin_size_out);
    if (bin_num < 10)
    {
        *bin_size_out = data_size / 10;
        bin_num = 10;
    }

    double *bin_data_temp = new double[bin_num];
    double *data_temp = new double[*bin_size_out];

    for (int i = 0; i < bin_num; i++)
    {
        std::copy(data + i * (*bin_size_out), data + (i + 1) * (*bin_size_out), data_temp);
        bin_data_temp[i] = sum<double>(data_temp, (*bin_size_out)) / (*bin_size_out);
    }

    double total = sum<double>(bin_data_temp, bin_num);

    for (int i = 0; i < bin_num; i++)
    {
        bin_data[i] = (total - bin_data_temp[i]) / (bin_num - 1);
    }

    delete[] bin_data_temp;
    delete[] data_temp;
}

void make_bin(int *data, int bin_size_in, int data_size, int *bin_data, int *bin_size_out)
{
    *bin_size_out = bin_size_in;
    int bin_num = data_size / (*bin_size_out);
    if (bin_num < 10)
    {
        *bin_size_out = data_size / 10;
        bin_num = 10;
    }

    int *bin_data_temp = new int[bin_num];
    int *data_temp = new int[*bin_size_out];

    for (int i = 0; i < bin_num; i++)
    {
        std::copy(data + i * (*bin_size_out), data + (i + 1) * (*bin_size_out), data_temp);
        bin_data_temp[i] = sum<int>(data_temp, (*bin_size_out)) / (*bin_size_out);
    }

    double total = sum<int>(bin_data_temp, bin_num);

    for (int i = 0; i < bin_num; i++)
    {
        bin_data[i] = (total - bin_data_temp[i]) / (bin_num - 1);
    }

    delete[] bin_data_temp;
    delete[] data_temp;
}

void square(double *arr1, double *arr2, int N)
{
    for (int i = 0; i < N; i++)
    {
        arr2[i] = arr1[i] * arr1[i];
    }
}

void square(int *arr1, int *arr2, int N)
{
    for (int i = 0; i < N; i++)
    {
        arr2[i] = arr1[i] * arr1[i];
    }
}

template <typename T>
void Jackknife(T *data, int bin_size, void (*func)(T *, T *, T *, int), T *data2, int data_size, double *average, double *error)
{
    int bin_size_out;
    T *bin_data = (T *)malloc((data_size / bin_size) * sizeof(T));
    T *bin_data2 = (T *)malloc((data_size / bin_size) * sizeof(T));
    T *f_result = (T *)malloc((data_size / bin_size) * sizeof(T));
    T *f_result2 = (T *)malloc((data_size / bin_size) * sizeof(T));

    make_bin(data, bin_size, data_size, bin_data, &bin_size_out);
    make_bin(data2, bin_size, data_size, bin_data2, &bin_size_out);
    func(bin_data, bin_data2, f_result, (data_size / bin_size));
    *average = sum<T>(f_result, (data_size / bin_size)) / (data_size / bin_size);
    square(f_result, f_result2, (data_size / bin_size));
    *error = sqrt((sum<T>(f_result2, (data_size / bin_size)) / (data_size / bin_size) - (*average) * (*average)) * ((data_size / bin_size) - 1));

    free(bin_data);
    free(bin_data2);
    free(f_result);
    free(f_result2);
}

template <typename T>
void Jackknife(T *data, int bin_size, int data_size, double *average, double *error)
{
    int bin_size_out;
    T *bin_data = (T *)malloc((data_size / bin_size) * sizeof(T));
    T *bin_data2 = (T *)malloc((data_size / bin_size) * sizeof(T));

    make_bin(data, bin_size, data_size, bin_data, &bin_size_out);

    *average = sum<T>(bin_data, (data_size / bin_size)) / (data_size / bin_size);
    square(bin_data, bin_data2, (data_size / bin_size));
    *error = sqrt((sum<T>(bin_data2, (data_size / bin_size)) / (data_size / bin_size) - (*average) * (*average)) * ((data_size / bin_size) - 1));

    free(bin_data);
    free(bin_data2);
}

void save_txt(double *array, int N, std::string name)
{
    std::ofstream file(name + ".txt");
    if (file.is_open())
    {
        for (int i = 0; i < N; ++i)
        {
            file << array[i] << '\n';
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}

int main(int argc, char *argv[])
{
    double *mag;
    double *mag2;
    double *mag4;
    double *mag_abs;
    double *ene;
    double *ene2;
    double Tc = 2.0 / log(1.0 + sqrt(2.0));

    int L = 16;
    int observation = 50000;
    int thermalization = 10000;
    double T = Tc;
    double h = 0.0;
    int seed = 111;

    int N = L * L;
    mag = (double *)malloc(observation * sizeof(double));
    mag2 = (double *)malloc(observation * sizeof(double));
    mag4 = (double *)malloc(observation * sizeof(double));
    mag_abs = (double *)malloc(observation * sizeof(double));
    ene = (double *)malloc(observation * sizeof(double));
    ene2 = (double *)malloc(observation * sizeof(double));

    std::cout << "## Algorithm = Metropolis\n";
    std::cout << "## L = " << L << '\n';
    std::cout << "## T = " << T << '\n';
    std::cout << "## h = " << h << '\n';
    std::cout << "## random seed = " << seed << '\n';
    std::cout << "## thermalization steps = " << thermalization << '\n';
    std::cout << "## observation steps = " << observation << '\n';

    MC(L, T, h, thermalization, mag, mag2, mag4, mag_abs, ene, ene2, observation, seed);

    save_txt(mag, observation, "mag");
    save_txt(mag2, observation, "mag2");
    save_txt(mag4, observation, "mag4");
    save_txt(mag_abs, observation, "mag_abs");
    save_txt(ene, observation, "ene");
    save_txt(ene2, observation, "ene2");

    double E, E_err;
    Jackknife<double>(ene, max(100, observation / 100), observation, &E, &E_err);
    double E2, E2_err;
    Jackknife<double>(ene2, max(100, observation / 100), observation, &E2, &E2_err);
    double M, M_err;
    Jackknife<double>(mag, max(100, observation / 100), observation, &M, &M_err);
    double M2, M2_err;
    Jackknife<double>(mag2, max(100, observation / 100), observation, &M2, &M2_err);
    double M4, M4_err;
    Jackknife<double>(mag4, max(100, observation / 100), observation, &M4, &M4_err);
    double C, C_err;
    Jackknife<double>(ene, max(100, observation / 100), variance, ene2, observation, &C, &C_err);
    C *= N / (T * T);
    C_err *= N / (T * T);
    double b, b_err;
    Jackknife<double>(mag2, max(100, observation / 100), binder, mag4, observation, &b, &b_err);

    std::cout << "### Outputs with errors estimated by Jackknife method ###" << '\n';
    std::cout << "T = " << T << '\n';
    std::cout << "Energy = " << E << " +- " << E_err << '\n';
    std::cout << "Energy^2 = " << E2 << " +- " << E2_err << '\n';
    std::cout << "Magnetization = " << M << " +- " << M_err << '\n';
    std::cout << "Magnetization^2 = " << M2 << " +- " << M2_err << '\n';
    std::cout << "Magnetization^4 = " << M4 << " +- " << M4_err << '\n';
    std::cout << "Specific heat = " << C << " +- " << C_err << '\n';
    std::cout << "Susceptibility = " << M2 / T * N << " +- " << M2_err / T * N << '\n';
    std::cout << "Binder ratio = " << b << " +- " << b_err << '\n';
}
